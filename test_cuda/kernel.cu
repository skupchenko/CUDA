// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/driver_types.h"

#include <stdio.h>


__global__ void kernel(long long * numElements)
{
	long long i = 0;
	while(i < 1000000000)
	{
		++i;
	}
	*numElements = i;
}

int main(void)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Allocate the host input numElements
	long long * h_numElements = (long long *)malloc(sizeof(long long));

	* h_numElements = 0;

	// Allocate the device input numElements
	long long * d_numElements = NULL;
	err = hipMalloc((void **) & d_numElements, sizeof(long long));

	err = hipMemcpy(d_numElements, h_numElements, sizeof(long long), hipMemcpyHostToDevice);

	kernel<<<1, 16>>>(d_numElements);

	//����� event'�
	hipEvent_t syncEvent;

	hipEventCreate(& syncEvent);		//������� event
	hipEventRecord(syncEvent, 0);		//���������� event
	hipEventSynchronize(syncEvent);	//�������������� event

	err = hipMemcpy(h_numElements, d_numElements, sizeof(long long), hipMemcpyDeviceToHost);

	printf("[%d elements]\n", *h_numElements);

	system("pause");

	return 0;

}
