// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/driver_types.h"

#include <stdio.h>


__global__ void kernel(long long * numElements)
{
	long long i = 0;
	while(i < 48000000)
	{
		++i;
	}
	*numElements = i;
}

int main(void)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Allocate the host input numElements
	long long * h_numElements = (long long *)malloc(sizeof(long long));

	* h_numElements = 0;

	// Allocate the device input numElements
	long long * d_numElements = NULL;
	err = hipMalloc((void **) & d_numElements, sizeof(long long));
	
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device numElements (error code %s)!\n", hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}

	// Copy the host h_numElements in host memory to the device input d_numElements in
	// device memory
	err = hipMemcpy(d_numElements, h_numElements, sizeof(long long), hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy h_numElements from host to device (error code %s)!\n", hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}

	kernel<<<1, 16>>>(d_numElements);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}

	//Handle event'�
	hipEvent_t syncEvent;

	hipEventCreate(& syncEvent);		//Create event
	hipEventRecord(syncEvent, 0);		//Record event
	hipEventSynchronize(syncEvent);	//Synchronize event

	// Copy the device result in device memory to the host result
	// in host memory.
	err = hipMemcpy(h_numElements, d_numElements, sizeof(long long), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy numElements from device to host (error code %s)!\n", hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}

	printf("[%d elements]\n", *h_numElements);

	// Free device global memory
	err = hipFree(d_numElements);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device d_numElements (error code %s)!\n", hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_numElements);

	// Reset the device and exit
	// cudaDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling cudaDeviceReset causes all profile data to be
	// flushed before the application exits
	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}

	system("pause");

	return 0;
}
