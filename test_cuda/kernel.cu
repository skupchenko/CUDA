
#include <hip/hip_runtime.h>
__global__ void kernel()
{
	long long i = 0;
	while(i < 10000000000)
	{
		++i;
	}
}

int main(void)
{

	kernel<<<1, 16>>>();

	//����� event'�
	hipEvent_t syncEvent;

	hipEventCreate(& syncEvent);		//������� event
	hipEventRecord(syncEvent, 0);		//���������� event
	hipEventSynchronize(syncEvent);	//�������������� event

	return 0;

}
